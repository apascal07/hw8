#include "hip/hip_runtime.h"
/**
* University of Pittsburgh
* Department of Computer Science
* CS1645: Introduction to HPC Systems
* Instructor Bryan Mills, PhD
* This is a skeleton for implementing prefix sum using GPU, inspired
* by nvidia course of similar name.
*/

#include <stdio.h>
#include "timer.h"
#include <math.h>
#include <string.h>

#define N 512

/*
* You should implement the simple scan function here!
*/
__global__ void scan_simple(float *g_odata, float *g_idata, int n) {
extern  __shared__  float x[];

// STUDENT: YOUR CODE GOES HERE.

/* Pseudo-code
1: for d = 1 to log2 n do
2:     for all k in parallel do
3:         if k >= 2^d  then
4:             x[out][k] = x[in][k – 2^(d-1)] + x[in][k]
5:         else
6:             x[out][k] = x[in][k]
*/

g_odata[threadIdx.x] = 0.0;


int thread_id = threadIdx.x;
int pout = 0, pin = 1;
// Load input into shared memory.
// This is exclusive scan, so shift right by one
// and set first element to 0
if (thread_id > 0) {
x[pout*n + thread_id] = g_idata[thread_id-1];
}
else {
x[pout*n + thread_id] = 0;
}
__syncthreads();
for (int offset = 1; offset < n; offset *= 2) {
pout = 1 - pout; // swap double buffer indices
pin = 1 - pout;
if (thread_id >= offset) {
x[pout*n+thread_id] = x[pin*n+thread_id - offset] + x[pin*n+thread_id];
}
else {
x[pout*n+thread_id] = x[pin*n+thread_id];
}
__syncthreads();
}
g_odata[thread_id] = x[pout*n+thread_id]; // write output

}

/*
* You should implement the prescan kernel function here!
*/
__global__ void prescan(float *g_odata, float *g_idata, int n) {
extern  __shared__  float x[];

// STUDENT: YOUR CODE GOES HERE.
/* Pseudo-code
1: for d = 0 to log2 n – 1 do
2:     for all k = 0 to n – 1 by 2 d+1 in parallel do
3:         x[k +  2^(d+1) – 1] = x[k +  2^d – 1] + x[k +  2^(d+1) – 1]
4:		   x[n – 1] <-- 0
5:		   for d = log2 n – 1 down to 0 do
6:             for all k = 0 to n – 1 by 2 d +1 in parallel do
7:             t = x[k +  2^d  – 1]
8:             x[k +  2^d – 1] = x[k +  2^(d+1) – 1]
9:             x[k +  2^(d+1) – 1] = t +  x[k +  2^(d+1) – 1]
*/
int thread_id = threadIdx.x;
int offset = 1;
x[2*thread_id] = g_idata[2*thread_id]; // load input into shared memory
x[2*thread_id+1] = g_idata[2*thread_id+1];
for (int d_reduction = n>>1; d_reduction > 0; d_reduction >>= 1) {  // build sum in place up the tree
__syncthreads();
if (thread_id < d_reduction) {
int ai = offset*(2*thread_id+1)-1;
int bi = offset*(2*thread_id+2)-1;
x[bi] += x[ai];
}
offset *= 2;
if (thread_id == 0) {
x[n - 1] = 0; // clear the last element
}
for (int d_down_sweep = 1; d_down_sweep < n; d_down_sweep *= 2) {  // traverse down tree & build scan
offset >>= 1;
__syncthreads();
if (thread_id < d_down_sweep) {
int ai = offset*(2*thread_id+1)-1;
int bi = offset*(2*thread_id+2)-1;
float t = x[ai];
x[ai] = x[bi];
x[bi] += t;
}
}
__syncthreads();
g_odata[2*thread_id] = x[2*thread_id]; // write results to device memory
g_odata[2*thread_id+1] = x[2*thread_id+1];
}
}

/*
* Fills an array a with n random floats.
*/
void random_floats(float* a, int n) {
float d;
// Comment out this line if you want consistent "random".
srand(time(NULL));
for (int i = 0; i < n; ++i) {
d = rand() % 8;
a[i] = ((rand() % 64) / (d > 0 ? d : 1));
}
}

/*
* Simple Serial implementation of scan.
*/
void serial_scan(float* out, float* in, int n) {
float total_sum = 0;
out[0] = 0;
for (int i = 1; i < n; i++) {
total_sum += in[i-1];
out[i] = out[i-1] + in[i-1];
}
if (total_sum != out[n-1]) {
printf("Warning: exceeding accuracy of float.\n");
}
}

/*
* This is a simple function that confirms that the output of the scan
* function matches that of a golden image (array).
*/
bool printError(float *gold_out, float *test_out, bool show_all) {
bool firstFail = true;
bool error = false;
float epislon = 0.1;
float diff = 0.0;
for (int i = 0; i < N; ++i) {
diff = abs(gold_out[i] - test_out[i]);
if ((diff > epislon) && firstFail) {
printf("ERROR: gold_out[%d] = %f != test_out[%d] = %f // diff = %f \n", i, gold_out[i], i, test_out[i], diff);
firstFail = show_all;
error = true;
}
}
return error;
}

int main(void) {
float *in, *out, *gold_out; // host
float *d_in, *d_out; // device
int size = sizeof(float) * N;

timerStart();
hipMalloc((void **)&d_in, size);
hipMalloc((void **)&d_out, size);

in = (float *)malloc(size);
random_floats(in, N);
out = (float *)malloc(size);
gold_out = (float *)malloc(size);
printf("TIME: Init took %d ms\n",  timerStop());
// ***********
// RUN SERIAL SCAN
// ***********
timerStart();
serial_scan(gold_out, in, N);
printf("TIME: Serial took %d ms\n",  timerStop());

timerStart();
hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
printf("TIME: Copy took %d ms\n",  timerStop());
// ***********
// RUN SIMPLE SCAN
// ***********
timerStart();
scan_simple<<< 1, 512, N * 2 * sizeof(float)>>>(d_out, d_in, N);
hipDeviceSynchronize();
printf("TIME: Simple kernel took %d ms\n",  timerStop());
timerStart();
hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
printf("TIME: Copy back %d ms\n",  timerStop());

if (printError(gold_out, out, false)) {
printf("ERROR: The simple scan function failed to produce proper output.\n");
} else {
printf("CONGRATS: The simple scan function produced proper output.\n");
}

// ***********
// RUN PRESCAN
// note size change in number of threads, only need 256 because each
// thread should handle 2 elements.
// ***********
timerStart();
prescan<<< 1, 256, N * 2 * sizeof(float)>>>(d_out, d_in, N);
hipDeviceSynchronize();
printf("TIME: Prescan kernel took %d ms\n",  timerStop());
timerStart();
hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
printf("TIME: Copy back %d ms\n",  timerStop());

if (printError(gold_out, out, false)) {
printf("ERROR: The prescan function failed to produce proper output.\n");
} else {
printf("CONGRATS: The prescan function produced proper output.\n");
}

return 0;
}